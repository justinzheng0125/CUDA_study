#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void helloCUDA(void) {

	printf("Hello CUDA from GPU\n");
}

int main2(void) {
	printf("Hellow GPU from CPU\n");
	helloCUDA <<< 1, 10 >>>();
	return 0;
}