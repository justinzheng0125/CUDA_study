#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string.h>
//#include <stblib.h>

// define size of the vector
#define NUM_DATA 512 //10240

__global__ void vecAdd(int* a, int* b, int* c)
{
	int tID = threadIdx.x;
	c[tID] = a[tID] + b[tID];
	//printf("%d + %d = %d\n", a[tID], b[tID], c[tID]);
}

__global__ void printing(int* a)
{
	int tID = threadIdx.x;
	printf("%d\n", a[tID]);
}

int main(void)
{
	int* a, * b, * c, * host_c;
	int* dev_a, * dev_b, * dev_c;

	int memSize = sizeof(int) * NUM_DATA;
	printf("%d elements, %d bytes memSize\n", NUM_DATA, memSize);

	a = new int[NUM_DATA]; memset(a, 0, memSize);
	b = new int[NUM_DATA]; memset(b, 0, memSize);
	c = new int[NUM_DATA]; memset(c, 0, memSize);
	host_c = new int[NUM_DATA]; memset(host_c, 0, memSize);

	for (int i = 0; i < NUM_DATA; i++)
	{
		a[i] = /*rand() % 10*/ i;
		b[i] = /*rand() % 10*/ i;
	}

	for (int i = 0; i < NUM_DATA; i++)
	{
		c[i] = a[i] + b[i];
	}

	hipMalloc(&dev_a, memSize);
	hipMalloc(&dev_b, memSize);
	hipMalloc(&dev_c, memSize);

	// Copy CPU data to GPU
	hipMemcpy(dev_a, a, memSize, hipMemcpyHostToDevice);
	//printing << <1, NUM_DATA >> > (dev_a);
	hipMemcpy(dev_b, b, memSize, hipMemcpyHostToDevice);

	// GPU computing
	vecAdd << <1, NUM_DATA >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	//printing << <1, NUM_DATA >> > (dev_c);

	// Copy result from GPU to CPU
	hipMemcpy(host_c, dev_c, memSize, hipMemcpyDeviceToHost);
	//for (int i = 0; i < NUM_DATA; i++) printf("%d\n", host_c[i]);

	bool res = true;
	for (int i = 0; i < NUM_DATA; i++)
	{
		if (host_c[i] != c[i])
		{
			printf("[%d] the result %d != %d\n", i, c[i], host_c[i]);
			res = false;
		}
	}
	if (res) printf("GPU works well\n");
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
	delete[] a; delete[] b; delete[] c;

	return 0;
}