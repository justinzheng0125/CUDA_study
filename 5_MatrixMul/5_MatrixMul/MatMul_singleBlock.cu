#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ROW_SIZE (32)
#define COL_SIZE (32)
#define K_SIZE (128)

#define WORK_LOAD (1024)
#define MAT_SIZE_A (ROW_SIZE*K_SIZE)
#define MAT_SIZE_B (K_SIZE*COL_SIZE)
#define MAT_SIZE_C (ROW_SIZE*COL_SIZE)

float A[ROW_SIZE][K_SIZE];
float B[K_SIZE][COL_SIZE];
float hostC[ROW_SIZE][COL_SIZE];
float deviceC[ROW_SIZE][COL_SIZE];

#define memSetZero(P, type, size) memset(P, 0, sizeof(type)*size)
//#define dMemAlloc(P, type, size) cudaMemAlloc(&P, sizeof(type)*size)

__global__ void mult_kernel(float* _A, float* _B, float* _C) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	int index = row * blockDim.x + col;
	if (row >= blockDim.y || col >= blockDim.x) {
		return;
	}
	for (int k = 0; k < K_SIZE; k++) {
		for (int i = 0; i < WORK_LOAD; i++) {
			_C[index] += _A[row * K_SIZE + k] * _B[k * COL_SIZE + col];
		}
	}
	
}

void main(void) {
	float* dA, * dB, * dC;
	dA = dB = dC = NULL;

	memSetZero(A, float, MAT_SIZE_A);
	memSetZero(B, float, MAT_SIZE_B);
	memSetZero(hostC, float, MAT_SIZE_C);
	memSetZero(deviceC, float, MAT_SIZE_C);

	hipMalloc(&dA, sizeof(float) * MAT_SIZE_A);
	hipMalloc(&dB, sizeof(float) * MAT_SIZE_B);
	hipMalloc(&dC, sizeof(float) * MAT_SIZE_C);

	// Generate Data
	for (int i = 0; i < ROW_SIZE; i++) {
		for (int j = 0; j < K_SIZE; j++) {
			A[i][j] = rand() % 100;
		}
	}
	for (int i = 0; i < K_SIZE; i++) {
		for (int j = 0; j < COL_SIZE; j++) {
			B[i][j] = rand() % 100;
		}
	}

	// Calculate Data - CPU
	for (int r = 0; r < ROW_SIZE; r++) {
		for (int c = 0; c < COL_SIZE; c++) {
			for (int k = 0; k < K_SIZE; k++) {
				for (int i = 0; i < WORK_LOAD; i++) {
					hostC[r][c] += A[r][k] * B[k][c];
				}
			}
		}
	}

	// copy Data to GPU
	hipMemcpy(dA, A, sizeof(float) * MAT_SIZE_A, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(float) * MAT_SIZE_B, hipMemcpyHostToDevice);

	dim3 blockDim(COL_SIZE, ROW_SIZE);
	mult_kernel << <1, blockDim >> > (dA, dB, dC);
	hipDeviceSynchronize();
	hipMemcpy(deviceC, dC, sizeof(float) * MAT_SIZE_C, hipMemcpyDeviceToHost);

	bool isCorrect = true;
	for (int i = 0; i < ROW_SIZE; i++) {
		for (int j = 0; j < COL_SIZE; j++) {
			if (hostC[i][j] != deviceC[i][j]) {
				isCorrect = false;
				break;
			}
		}
	}
	if (isCorrect) printf("working Correct!!\n");
	else printf("result is WRONG!!\n");
}

